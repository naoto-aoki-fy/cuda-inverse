// inv_cublas_z.cu
// nvcc -gencode=arch=compute_61,code=sm_61 main_cublas.cu -lcublas
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

#define CHECK_CUDA(call) \
  do { hipError_t e = (call); if (e != hipSuccess) { \
    fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); exit(EXIT_FAILURE); } } while(0)

#define CHECK_CUBLAS(call) \
  do { hipblasStatus_t s = (call); if (s != HIPBLAS_STATUS_SUCCESS) { \
    fprintf(stderr, "cuBLAS error %s:%d: %d\n", __FILE__, __LINE__, (int)s); exit(EXIT_FAILURE); } } while(0)

// print column-major cuDoubleComplex matrix (n x n)
void printMat(const hipDoubleComplex* A, int n){
    for(int i=0;i<n;i++){
      for(int j=0;j<n;j++){
        hipDoubleComplex v = A[i + j*n]; // column-major
        printf("(%9.5f,%9.5f) ", v.x, v.y);
      }
      printf("\n");
    }
    printf("\n");
}

int main(){
  const int n = 3;                 // 行列サイズ n x n
  const int lda = n;
  const int batchSize = 1;         // 単一行列を batched で処理

  // --- ホスト側：サンプル行列（列優先 column-major） ---
  hipDoubleComplex h_A[n*n];
  // 例: A = [[1+i, 2, 3], [4, 5+i, 6], [7, 8, 9+i]] を列優先で格納
  h_A[0 + 0*n] = make_hipDoubleComplex(1.0, 1.0); // A(0,0)
  h_A[1 + 0*n] = make_hipDoubleComplex(4.0, 0.0); // A(1,0)
  h_A[2 + 0*n] = make_hipDoubleComplex(7.0, 0.0); // A(2,0)

  h_A[0 + 1*n] = make_hipDoubleComplex(2.0, 0.0); // A(0,1)
  h_A[1 + 1*n] = make_hipDoubleComplex(5.0, 1.0); // A(1,1)
  h_A[2 + 1*n] = make_hipDoubleComplex(8.0, 0.0); // A(2,1)

  h_A[0 + 2*n] = make_hipDoubleComplex(3.0, 0.0); // A(0,2)
  h_A[1 + 2*n] = make_hipDoubleComplex(6.0, 0.0); // A(1,2)
  h_A[2 + 2*n] = make_hipDoubleComplex(9.0, 1.0); // A(2,2)

  printf("Host A (column-major):\n");
  printMat(h_A, n);

  // --- デバイスメモリ確保 ---
  hipDoubleComplex* d_A = nullptr;        // 作業用（LU→inverse）
  hipDoubleComplex* d_Aorig = nullptr;    // 元の行列を保持（積計算に使う）
  hipDoubleComplex* d_C = nullptr;        // 積の結果 (Aorig * Ainv)
  CHECK_CUDA(hipMalloc((void**)&d_A, sizeof(hipDoubleComplex)*n*n));
  CHECK_CUDA(hipMalloc((void**)&d_Aorig, sizeof(hipDoubleComplex)*n*n));
  CHECK_CUDA(hipMalloc((void**)&d_C, sizeof(hipDoubleComplex)*n*n));

  // コピー: d_A にコピーしてその場で LU/逆行列計算をする（in-place）
  CHECK_CUDA(hipMemcpy(d_A, h_A, sizeof(hipDoubleComplex)*n*n, hipMemcpyHostToDevice));
  // d_Aorig にもコピーして、inverse と掛ける時に使う
  CHECK_CUDA(hipMemcpy(d_Aorig, h_A, sizeof(hipDoubleComplex)*n*n, hipMemcpyHostToDevice));

  // デバイス上の行列ポインタ配列（batched API用）
  hipDoubleComplex* h_Aptr[batchSize];
  h_Aptr[0] = d_A;
  hipDoubleComplex** d_Aptr = nullptr;
  CHECK_CUDA(hipMalloc((void**)&d_Aptr, sizeof(hipDoubleComplex*) * batchSize));
  CHECK_CUDA(hipMemcpy(d_Aptr, h_Aptr, sizeof(hipDoubleComplex*) * batchSize, hipMemcpyHostToDevice));

  // pivots と info（デバイス上）
  int* d_pivots = nullptr;   // pivot array: length = n * batchSize
  int* d_info   = nullptr;   // info array: length = batchSize
  CHECK_CUDA(hipMalloc((void**)&d_pivots, sizeof(int) * n * batchSize));
  CHECK_CUDA(hipMalloc((void**)&d_info,   sizeof(int) * batchSize));

  // --- cuBLAS ハンドル ---
  hipblasHandle_t handle;
  CHECK_CUBLAS(hipblasCreate(&handle));

  // LU 分解（in-place）
  CHECK_CUBLAS(hipblasZgetrfBatched(handle, n, (hipDoubleComplex**)d_Aptr, lda, d_pivots, d_info, batchSize));

  // info をホストに戻してエラー確認
  int h_info[batchSize];
  CHECK_CUDA(hipMemcpy(h_info, d_info, sizeof(int)*batchSize, hipMemcpyDeviceToHost));
  for(int i=0;i<batchSize;i++){
    if(h_info[i] != 0){
      fprintf(stderr, "getrfBatched failed for batch %d: info=%d\n", i, h_info[i]);
      // cleanup before exit
      hipblasDestroy(handle);
      hipFree(d_Aptr);
      hipFree(d_A);
      hipFree(d_Aorig);
      hipFree(d_pivots);
      hipFree(d_info);
      hipFree(d_C);
      exit(EXIT_FAILURE);
    }
  }

  // 逆行列計算（in-place: d_A が逆行列に置き換わる）
  // 注意: getriBatched のシグネチャは CUDA バージョンによって差があるので
  // コンパイル時にヘッダを確認してください。ここでは一般的な形を使います。
  CHECK_CUBLAS(hipblasZgetriBatched(handle, n, (hipDoubleComplex**)d_Aptr, lda, d_pivots, (hipDoubleComplex**)d_Aptr, lda, d_info, batchSize));

  // info 再チェック
  CHECK_CUDA(hipMemcpy(h_info, d_info, sizeof(int)*batchSize, hipMemcpyDeviceToHost));
  for(int i=0;i<batchSize;i++){
    if(h_info[i] != 0){
      fprintf(stderr, "getriBatched failed for batch %d: info=%d\n", i, h_info[i]);
      // cleanup before exit
      hipblasDestroy(handle);
      hipFree(d_Aptr);
      hipFree(d_A);
      hipFree(d_Aorig);
      hipFree(d_pivots);
      hipFree(d_info);
      hipFree(d_C);
      exit(EXIT_FAILURE);
    }
  }

  // d_A は逆行列になっている。これを d_Ainv と見なす。
  hipDoubleComplex* d_Ainv = d_A;

  // --- 元行列 * 逆行列 を計算 (C = Aorig * Ainv) ---
  hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
  hipDoubleComplex beta  = make_hipDoubleComplex(0.0, 0.0);

  // cublasZgemm(handle, transA, transB, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc)
  CHECK_CUBLAS(hipblasZgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_N,
                           n, n, n,
                           &alpha,
                           d_Aorig, lda,
                           d_Ainv,  lda,
                           &beta,
                           d_C,     lda));

  // 結果をホストにコピーして表示
  hipDoubleComplex h_inv[n*n];
  hipDoubleComplex h_prod[n*n];
  CHECK_CUDA(hipMemcpy(h_inv, d_Ainv, sizeof(hipDoubleComplex)*n*n, hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(h_prod, d_C, sizeof(hipDoubleComplex)*n*n, hipMemcpyDeviceToHost));

  printf("Inverse (from device):\n");
  printMat(h_inv, n);

  printf("Product A * Ainv (should be identity):\n");
  printMat(h_prod, n);

  // 誤差評価（最大ノルム）
  double max_err = 0.0;
  for(int i=0;i<n;i++){
    for(int j=0;j<n;j++){
      double expect_re = (i==j) ? 1.0 : 0.0;
      double expect_im = 0.0;
      hipDoubleComplex v = h_prod[i + j*n];
      double dr = v.x - expect_re;
      double di = v.y - expect_im;
      double abs_err = sqrt(dr*dr + di*di);
      if(abs_err > max_err) max_err = abs_err;
    }
  }
  printf("max abs error vs I: %.6e\n", max_err);

  // cleanup
  CHECK_CUBLAS(hipblasDestroy(handle));
  CHECK_CUDA(hipFree(d_Aptr));
  CHECK_CUDA(hipFree(d_A));
  CHECK_CUDA(hipFree(d_Aorig));
  CHECK_CUDA(hipFree(d_pivots));
  CHECK_CUDA(hipFree(d_info));
  CHECK_CUDA(hipFree(d_C));

  return 0;
}
