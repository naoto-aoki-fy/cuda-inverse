// invert_complex_check_fixed.cu
// nvcc -gencode=arch=compute_61,code=sm_61 main_cusolver.cu -lcusolver -lcublas
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

#define CHECK_CUDA(call) { \
  hipError_t e = (call); \
  if (e != hipSuccess) { \
    fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
    exit(EXIT_FAILURE); \
  } \
}
#define CHECK_CUSOLVER(call) { \
  hipsolverStatus_t s = (call); \
  if (s != HIPSOLVER_STATUS_SUCCESS) { \
    fprintf(stderr, "cuSolver Error %s:%d: %d\n", __FILE__, __LINE__, (int)s); \
    exit(EXIT_FAILURE); \
  } \
}
#define CHECK_CUBLAS(call) { \
  hipblasStatus_t s = (call); \
  if (s != HIPBLAS_STATUS_SUCCESS) { \
    fprintf(stderr, "cuBLAS Error %s:%d: %d\n", __FILE__, __LINE__, (int)s); \
    exit(EXIT_FAILURE); \
  } \
}

int main() {
    const int n = 3;
    const int lda = n;

    // column-major: columns concatenated
    hipDoubleComplex h_A[n*n] = {
        make_hipDoubleComplex(1.0,  2.0), // (0,0)
        make_hipDoubleComplex(0.0,  0.0), // (1,0)
        make_hipDoubleComplex(5.0, -3.0), // (2,0)

        make_hipDoubleComplex(2.0,  0.0), // (0,1)
        make_hipDoubleComplex(1.0,  0.0), // (1,1)
        make_hipDoubleComplex(6.0,  0.0), // (2,1)

        make_hipDoubleComplex(3.0, -1.0), // (0,2)
        make_hipDoubleComplex(4.0,  2.0), // (1,2)
        make_hipDoubleComplex(0.0,  1.0)  // (2,2)
    };

    std::vector<hipDoubleComplex> h_Ainv(n*n);
    std::vector<hipDoubleComplex> h_prod(n*n);

    // handles
    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));
    CHECK_CUBLAS(hipblasCreate(&cublasH));

    // device buffers (initialize to nullptr for safe cleanup)
    hipDoubleComplex *d_A = nullptr, *d_Aorig = nullptr, *d_B = nullptr, *d_work = nullptr;
    hipDoubleComplex *d_prod = nullptr;
    int *d_ipiv = nullptr;
    int *d_info = nullptr;

    CHECK_CUDA(hipMalloc((void**)&d_A, sizeof(hipDoubleComplex)*n*n));
    CHECK_CUDA(hipMalloc((void**)&d_Aorig, sizeof(hipDoubleComplex)*n*n)); // keep original A
    CHECK_CUDA(hipMalloc((void**)&d_B, sizeof(hipDoubleComplex)*n*n)); // identity -> X (inverse)
    CHECK_CUDA(hipMalloc((void**)&d_ipiv, sizeof(int)*n));
    CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_prod, sizeof(hipDoubleComplex)*n*n));

    // copy A to device (both to d_A and d_Aorig)
    CHECK_CUDA(hipMemcpy(d_A, h_A, sizeof(hipDoubleComplex)*n*n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Aorig, h_A, sizeof(hipDoubleComplex)*n*n, hipMemcpyHostToDevice));

    // prepare identity on device (right-hand side)
    std::vector<hipDoubleComplex> h_I(n*n);
    for (int i=0;i<n*n;++i) h_I[i] = make_hipDoubleComplex(0.0, 0.0);
    for (int i=0;i<n;++i) h_I[i + i*n] = make_hipDoubleComplex(1.0, 0.0); // column-major: i + i*n
    CHECK_CUDA(hipMemcpy(d_B, h_I.data(), sizeof(hipDoubleComplex)*n*n, hipMemcpyHostToDevice));

    // getrf buffer size (Z = double complex)
    int lwork = 0;
    CHECK_CUSOLVER(hipsolverDnZgetrf_bufferSize(cusolverH, n, n, d_A, lda, &lwork));
    CHECK_CUDA(hipMalloc((void**)&d_work, sizeof(hipDoubleComplex)*lwork));

    // --- Important: declare variables that might be referenced after a goto BEFORE any goto ---
    hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    hipDoubleComplex beta  = make_hipDoubleComplex(0.0, 0.0);
    double max_err = 0.0;

    // LU (A -> LU)
    CHECK_CUSOLVER(hipsolverDnZgetrf(cusolverH, n, n, d_A, lda, d_work, d_ipiv, d_info));
    int info_h = 0;
    CHECK_CUDA(hipMemcpy(&info_h, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (info_h != 0) {
        fprintf(stderr, "getrf failed, info=%d (matrix may be singular)\n", info_h);
        goto CLEANUP;
    }

    // solve A * X = I -> X (inverse) stored in d_B
    CHECK_CUSOLVER(hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, n, n, d_A, lda, d_ipiv, d_B, n, d_info));
    CHECK_CUDA(hipMemcpy(&info_h, d_info, sizeof(int), hipMemcpyDeviceToHost));
    if (info_h != 0) {
        fprintf(stderr, "getrs failed, info=%d\n", info_h);
        goto CLEANUP;
    }

    // d_Aorig contains original A, d_B contains A^{-1}
    // Compute prod = A * Ainv using cuBLAS:
    // C = alpha * A * B + beta * C
    CHECK_CUBLAS(hipblasZgemm(cublasH,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             n, n, n,
                             &alpha,
                             (const hipDoubleComplex*)d_Aorig, lda,   // original A
                             (const hipDoubleComplex*)d_B, n,         // B = Ainv
                             &beta,
                             d_prod, n));                             // C (product)

    // Copy product back to host and compute max abs error vs identity
    CHECK_CUDA(hipMemcpy(h_prod.data(), d_prod, sizeof(hipDoubleComplex)*n*n, hipMemcpyDeviceToHost));

    max_err = 0.0;
    for (int row = 0; row < n; ++row) {
        for (int col = 0; col < n; ++col) {
            hipDoubleComplex z = h_prod[col*n + row]; // column-major
            double re = hipCreal(z);
            double im = hipCimag(z);
            double expect_re = (row == col) ? 1.0 : 0.0;
            double expect_im = 0.0;
            double dre = re - expect_re;
            double dim = im - expect_im;
            double abs_err = sqrt(dre*dre + dim*dim);
            if (abs_err > max_err) max_err = abs_err;
        }
    }

    // print product matrix
    printf("A * Ainv (row major display):\n");
    for (int row=0; row<n; ++row) {
        for (int col=0; col<n; ++col) {
            hipDoubleComplex z = h_prod[col*n + row];
            printf("% .6f%+.6fi  ", hipCreal(z), hipCimag(z));
        }
        printf("\n");
    }
    printf("max absolute error vs identity: %.6e\n", max_err);
    if (max_err < 1e-12) {
        printf("OK: result is numerically identity (tol 1e-12)\n");
    } else {
        printf("WARNING: not exact identity (max_err=%.6e)\n", max_err);
    }

CLEANUP:
    if (d_work) hipFree(d_work);
    if (d_prod) hipFree(d_prod);
    if (d_B) hipFree(d_B);
    if (d_Aorig) hipFree(d_Aorig);
    if (d_A) hipFree(d_A);
    if (d_ipiv) hipFree(d_ipiv);
    if (d_info) hipFree(d_info);
    if (cublasH) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    return 0;
}
