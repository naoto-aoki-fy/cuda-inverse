/* ----  STL-free core / STL in main only  ---- */
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>   // ← main 用
#include <omp.h>
#include <hip/hip_runtime.h>

/* 手書き swap（テンプレートを避ける） */
__device__
inline void swap_int (int&    a, int&    b) { int    t = a; a = b; b = t; }
__device__
inline void swap_dbl (double& a, double& b) { double t = a; a = b; b = t; }

/* ---------- LU 分解（部分ピボット・順次） ---------- */
__device__
void lu_decompose_seq(double* A, int* piv, int n)
{
    __shared__ double Akk;
    if (threadIdx.x==0) {
        for (int i = 0; i < n; ++i) piv[i] = i;
    }

    for (int k = 0; k < n; ++k) {
        if (threadIdx.x==0) {
            /* ピボット選択 */
            int    pivot = k;
            double amax  = fabs(A[k * n + k]);
            for (int i = k + 1; i < n; ++i) {
                double val = fabs(A[i * n + k]);
                if (val > amax) { amax = val; pivot = i; }
            }
            if (amax == 0.0) {
                return;
                // std::fprintf(stderr, "Singular\n"); std::exit(EXIT_FAILURE);
            }

            /* 行入れ替え */
            if (pivot != k) {
                swap_int(piv[k], piv[pivot]);
                for (int j = 0; j < n; ++j)
                    swap_dbl(A[k * n + j], A[pivot * n + j]);
            }

            /* 前進消去 */
            Akk = A[k * n + k];
        }
        __syncthreads();
        // #pragma omp parallel for schedule(static)
        // for (int i = k + 1; i < n; ++i) {
        for (int i = k + 1 + threadIdx.x; i < n; i += blockDim.x) {
            double const Lik = (A[i * n + k] /= Akk);
            for (int j = k + 1; j < n; ++j)
                A[i * n + j] -= Lik * A[k * n + j];
        // }
        }
    }
}

/* ---------- 前進・後退代入 ---------- */
__device__
void lu_solve(const double* LU,
              const int*    piv,
              const double* b,
              double*       y,
              double*       x,
              int           n)
{
    /* Pb → y */
    for (int i = 0; i < n; ++i) y[i] = b[piv[i]];

    /* Ly = Pb */
    for (int i = 0; i < n; ++i) {
        double sum = y[i];
        for (int j = 0; j <  i; ++j) sum -= LU[i * n + j] * y[j];
        y[i] = sum;
    }

    /* Ux = y */
    for (int i = n - 1; i >= 0; --i) {
        double sum = y[i];
        for (int j = i + 1; j < n; ++j) sum -= LU[i * n + j] * x[j];
        x[i] = sum / LU[i * n + i];
    }
}

/* ---------- 逆行列 ---------- */
__global__
void invert(const double* A,
            double*       Ainv,
            double*       LU,
            int*          piv,
            double*       b,
            double*       y,
            double*       x,
            int           n)
{
    /* 1. LU 分解 */
    // for (int i = 0; i < n * n; ++i) LU[i] = A[i];
    lu_decompose_seq(LU, piv, n);

    /* 2. n 本の方程式を解く（列ごとに単位ベクトル） */
    // #pragma omp parallel
    {
        // int tid = omp_get_thread_num();
        int const tid = threadIdx.x;
        double* b_loc = b + tid * n;
        double* y_loc = y + tid * n;
        double* x_loc = x + tid * n;

        // #pragma omp for schedule(static)
        // for (int col = 0; col < n; ++col) {
        for (int col = threadIdx.x; col < n; col += blockDim.x) {
            for (int i = 0; i < n; ++i) b_loc[i] = 0.0;
            b_loc[col] = 1.0;

            lu_solve(LU, piv, b_loc, y_loc, x_loc, n);

            for (int row = 0; row < n; ++row)
                Ainv[row * n + col] = x_loc[row];
        }
        // }
    }
}

/* ---------- テスト ---------- */
int main()
{
    const int n = 60;
    int block_size = 32;
    // const int threads = omp_get_max_threads();

    /* main だけで STL を使用して領域確保 */
    std::vector<double> A   (n * n);
    std::vector<double> LU  (n * n);
    std::vector<double> Ainv(n * n);

    /* テスト行列生成（乱数） */
    std::srand(1234);
    for (double& v : A) {
        v = ( -1.0 + 2.0 * (std::rand() / static_cast<double>(RAND_MAX)) );
    }

    

    double* A_device;
    hipMalloc(&A_device, n * n * sizeof(double));
    hipMemcpyAsync(A_device, A.data(), n * n * sizeof(double), hipMemcpyHostToDevice, 0);


    double* LU_device;
    hipMallocAsync(&LU_device, n * n * sizeof(double), 0);
    double* Ainv_device;
    hipMallocAsync(&Ainv_device, n * n * sizeof(double), 0);
    int* piv_device;
    hipMallocAsync(&piv_device, n * sizeof(int), 0);
    double* b_device;
    hipMallocAsync(&b_device, block_size * n * sizeof(double), 0);
    double* y_device;
    hipMallocAsync(&y_device, block_size * n * sizeof(double), 0);
    double* x_device;
    hipMallocAsync(&x_device, block_size * n * sizeof(double), 0);

    /* 逆行列計算 */

    hipMemcpyAsync(LU_device, A_device, n * n * sizeof(double), hipMemcpyDeviceToDevice);

    invert<<<1, block_size>>>(A_device, Ainv_device,
           LU_device, piv_device,
           b_device, y_device, x_device, n);

    hipMemcpyAsync(Ainv.data(), Ainv_device, n * n * sizeof(double), hipMemcpyDeviceToHost, 0);

    hipStreamSynchronize(0);

    /* Frobenius 誤差 ||I - A·A⁻¹||_F */
    double err2 = 0.0;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            double s = 0.0;
            for (int k = 0; k < n; ++k)
                s += A[i * n + k] * Ainv[k * n + j];
            double diff = s - (i == j ? 1.0 : 0.0);
            err2 += diff * diff;
        }
    }
    std::printf("Frobenius error = %.3e\n", std::sqrt(err2));
    return 0;
}
